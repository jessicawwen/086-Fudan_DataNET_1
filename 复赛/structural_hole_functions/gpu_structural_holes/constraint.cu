#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "common.h"
#define NODES_PER_BLOCK 1

namespace gpu_easygraph {

enum norm_t { SUM = 0, MAX = 1 };

static __device__ double mutual_weight(
    const int* V,
    const int* E,
    const double* W,
    int u,
    int v
) {
    double a_uv = 0.0;
    for (int i = V[u]; i < V[u+1]; i++) {
        if (E[i] == v) {
            a_uv = W[i];
            break;
        }
    }
    return a_uv;
}

static __device__ double normalized_mutual_weight(
    const int* V,
    const int* E,
    const double* W, 
    int u,
    int v,
    norm_t norm
) {
    double weight_uv = mutual_weight(V, E, W, u, v);

    double scale = 0.0;
    if (norm == SUM) {
        for (int i = V[u]; i < V[u+1]; i++) {
            int neighbor = E[i];
            double weight_uw = mutual_weight(V, E, W, u, neighbor);
            scale += weight_uw;
        }
    } else if (norm == MAX) {
        for (int i = V[u]; i < V[u+1]; i++) {
            int neighbor = E[i];
            double weight_uw = mutual_weight(V, E, W, u, neighbor);
            scale = fmax(scale,weight_uw);
        }
    }
    return (scale==0.0) ? 0.0 : (weight_uv / scale);
}

static __device__ double local_constraint(
    const int* V,
    const int* E,
    const double* W,
    int u,
    int v
) {
    double direct = normalized_mutual_weight(V,E,W,u,v,SUM);
    double indirect = 0.0;
    for (int i = V[u]; i < V[u+1]; i++) {
        int neighbor = E[i];
        double norm_uw = normalized_mutual_weight(V, E, W, u, neighbor,SUM);
        double norm_wv = normalized_mutual_weight(V, E, W, neighbor, v,SUM);
        indirect += norm_uw * norm_wv;
    }
    double local_constraint_of_uv = (direct + indirect) * (direct + indirect);
    return local_constraint_of_uv;
}

__global__ void calculate_constraints(
    const int* __restrict__ V,
    const int* __restrict__ E,
    const double* __restrict__ W, 
    const int num_nodes, 
    const int* __restrict__ node_mask,
    double* __restrict__ constraint_results
) {
    int start_node = blockIdx.x * NODES_PER_BLOCK;
    int end_node = min(start_node + NODES_PER_BLOCK, num_nodes);

    for (int v = start_node; v < end_node; ++v) {
        if (!node_mask[v]) continue;

        double constraint_of_v = 0.0;
        bool is_nan = true;

        __shared__ double shared_constraint[256];
        double local_sum = 0.0;

        for (int i = V[v] + threadIdx.x; i < V[v + 1]; i += blockDim.x) {
            is_nan = false;
            int neighbor = E[i];
            local_sum += local_constraint(V, E, W, v, neighbor);
        }

        shared_constraint[threadIdx.x] = local_sum;
        __syncthreads();

        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (threadIdx.x < offset) {
                shared_constraint[threadIdx.x] += shared_constraint[threadIdx.x + offset];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            constraint_results[v] = (is_nan) ? NAN : shared_constraint[0];
        }
    }
}

static __device__ double directed_mutual_weight(
    const int* V,
    const int* E,
    const double* W,
    int u,
    int v
) {
    double a_uv = 0.0, a_vu = 0.0;
    for (int i = V[u]; i < V[u+1]; i++) {
        if (E[i] == v) {
            a_uv = W[i];
            break;
        }
    }
    for (int i = V[v]; i < V[v+1]; i++) {
        if (E[i] == u) {
            a_vu = W[i];
            break;
        }
    }
    return a_uv + a_vu;
}

static __device__ double directed_normalized_mutual_weight(
    const int* V,
    const int* E,
    const int* row, 
    const int* col, 
    const double* W, 
    int num_edges,
    int u,
    int v,
    norm_t norm
) {
    double weight_uv = directed_mutual_weight(V, E, W, u, v);

    double scale = 0.0;
    if(norm==SUM){
        for (int i = V[u]; i < V[u+1]; i++) {
            int neighbor = E[i];
            double weight_uw = directed_mutual_weight(V, E, W, u, neighbor);
            scale += weight_uw;
        }

        for (int i = 0; i < num_edges; i++) {
            if (col[i] == u) {
                int neighbor = row[i];
                double weight_wu = directed_mutual_weight(V, E, W, u, neighbor);
                scale += weight_wu;
            }
        }
    }else if(norm==MAX){
        for (int i = V[u]; i < V[u+1]; i++) {
            int neighbor = E[i];
            double weight_uw = directed_mutual_weight(V, E, W, u, neighbor);
            scale = fmax(scale,weight_uw);
        }

        for (int i = 0; i < num_edges; i++) {
            if (col[i] == u) {
                int neighbor = row[i];
                double weight_wu = directed_mutual_weight(V, E, W, u, neighbor);
                scale = fmax(scale,weight_wu);
            }
        }
    }
    return (scale==0.0) ? 0.0 : (weight_uv / scale);
}

static __device__ double directed_local_constraint(
    const int* V,
    const int* E,
    const int* row, 
    const int* col, 
    const double* W,
    int num_edges,
    int u,
    int v
) {
    double direct = directed_normalized_mutual_weight(V,E,row,col,W,num_edges,u,v,SUM);
    double indirect = 0.0;
    for (int i = V[u]; i < V[u+1]; i++) {
        int neighbor = E[i];
        double norm_uw = directed_normalized_mutual_weight(V, E, row, col, W, num_edges, u, neighbor,SUM);
        double norm_wv = directed_normalized_mutual_weight(V, E, row, col, W, num_edges, neighbor, v,SUM);
        indirect += norm_uw * norm_wv;
    }

    for (int i = 0; i < num_edges; i++) {
        if (col[i] == u) {
            int neighbor = row[i];
            double norm_uw = directed_normalized_mutual_weight(V, E, row, col, W, num_edges, u, neighbor,SUM);
            double norm_wv = directed_normalized_mutual_weight(V, E, row, col, W, num_edges, neighbor, v,SUM);
            indirect += norm_uw * norm_wv;
        }
    }
    double local_constraint_of_uv = (direct + indirect) * (direct + indirect);
    return local_constraint_of_uv;
}

__global__ void directed_calculate_constraints(
    const int* V,
    const int* E,
    const int* row, 
    const int* col, 
    const double* W,  
    int num_nodes,
    int num_edges,
    int* node_mask,
    double* constraint_results
) {
    int start_node = blockIdx.x * NODES_PER_BLOCK;
    int end_node = min(start_node + NODES_PER_BLOCK, num_nodes);

    for (int v = start_node; v < end_node; ++v) {
        if (!node_mask[v]) continue;

        double constraint_of_v = 0.0;
        bool is_nan = true;

        __shared__ double shared_constraint[256];
        double local_sum = 0.0;

        for (int i = V[v] + threadIdx.x; i < V[v + 1]; i += blockDim.x) {
            is_nan = false;
            int neighbor = E[i];
            local_sum += directed_local_constraint(V, E, row, col, W, num_edges, v, neighbor);
        }

        for (int i = threadIdx.x; i < num_edges; i += blockDim.x) {
            if (col[i] == v) {
                // is_nan = false;
                int neighbor = row[i];
                local_sum += directed_local_constraint(V, E, row, col, W, num_edges, v, neighbor);
            }
        }

        shared_constraint[threadIdx.x] = local_sum;
        __syncthreads();

        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (threadIdx.x < offset) {
                shared_constraint[threadIdx.x] += shared_constraint[threadIdx.x + offset];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            constraint_results[v] = (is_nan) ? NAN : shared_constraint[0];
        }
    }
}


int cuda_constraint(
    _IN_ const int* V,
    _IN_ const int* E,
    _IN_ const int* row,
    _IN_ const int* col,
    _IN_ const double* W,
    _IN_ int num_nodes,
    _IN_ int num_edges,
    _IN_ bool is_directed,
    _IN_ int* node_mask,
    _OUT_ double* constraint_results
) {
    int cuda_ret = hipSuccess;
    int EG_ret = EG_GPU_SUCC;
    
    int* d_V;
    int* d_E;
    int* d_row;
    int* d_col;
    double* d_W;
    int* d_node_mask;
    double* d_constraint_results;
    int block_size = 256;
    int grid_size = (num_nodes + NODES_PER_BLOCK - 1) / NODES_PER_BLOCK;

    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_V, (num_nodes+1) * sizeof(int)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_E, num_edges * sizeof(int)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_row, num_edges * sizeof(int)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_col, num_edges * sizeof(int)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_W, num_edges * sizeof(double)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_node_mask, num_nodes * sizeof(int)));
    EXIT_IF_CUDA_FAILED(hipMalloc((void**)&d_constraint_results, num_nodes * sizeof(double)));

    EXIT_IF_CUDA_FAILED(hipMemcpy(d_V, V, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_E, E, num_edges * sizeof(int), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_row, row, num_edges * sizeof(int), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_col, col, num_edges * sizeof(int), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_node_mask, node_mask, num_nodes * sizeof(int), hipMemcpyHostToDevice));
    EXIT_IF_CUDA_FAILED(hipMemcpy(d_W, W, num_edges * sizeof(double), hipMemcpyHostToDevice));

    if(is_directed){
        directed_calculate_constraints<<<grid_size, block_size>>>(d_V, d_E, d_row, d_col, d_W, num_nodes, num_edges, d_node_mask, d_constraint_results);
    }else{
        calculate_constraints<<<grid_size, block_size>>>(d_V, d_E, d_W, num_nodes, d_node_mask, d_constraint_results);
    }

    EXIT_IF_CUDA_FAILED(hipMemcpy(constraint_results, d_constraint_results, num_nodes * sizeof(double), hipMemcpyDeviceToHost));
exit:

    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_W);
    hipFree(d_node_mask);
    hipFree(d_constraint_results);
    if (cuda_ret != hipSuccess) {
        switch (cuda_ret) {
            case hipErrorOutOfMemory:
                EG_ret = EG_GPU_FAILED_TO_ALLOCATE_DEVICE_MEM;
                break;
            default:
                EG_ret = EG_GPU_DEVICE_ERR;
                break;
        }
    }

    return EG_ret; 
}

} // namespace gpu_easygraph